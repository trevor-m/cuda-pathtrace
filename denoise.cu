#include "hip/hip_runtime.h"
#include "denoise.h"

union Color  // 4 bytes = 4 chars = 1 float
{
	float c;
	uchar4 components;
};

__global__ void denoise_kernel(OutputBuffer input, float* d_output) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= input.width || y >= input.height)
    return;

  float3 color = make_float3(input.buffer[x*input.width*14 + y*14 + 0], input.buffer[x*input.width*14 + y*14 + 1], input.buffer[x*input.width*14 + y*14 + 2]);
  // clamp to range 0-1
  color.x = min(max(color.x, 0.0f), 1.0f);
  color.y = min(max(color.y, 0.0f), 1.0f);
  color.z = min(max(color.z, 0.0f), 1.0f);

  Color formatColor;
  formatColor.components = make_uchar4((unsigned char)(color.x*255.0), (unsigned char)(color.y*255.0), (unsigned char)(color.z*255.0), 1);

  // x and y are mixed up because I modified the eye rays so that the image output would be correct...
  d_output[x*input.width*3 + y*3 + 0] = y;
  d_output[x*input.width*3 + y*3 + 1] = input.width - x;
  d_output[x*input.width*3 + y*3 + 2] = formatColor.c;
}