#include "hip/hip_runtime.h"
#include "Camera.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <hip/hip_vector_types.h>
#include <hip/hip_math_constants.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

// Dimensions
#define SCREEN_W 512
#define SCREEN_H 512
#define NUM_CHANNELS 3

// Renderer constants
#define SAMPLES 4096
#define MAX_BOUNCES 10
#define PUSH_RAY_ORIGIN 0.05f

#include <hip/hip_runtime_api.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct Sphere {
  float radius;
  float3 pos;
  
  // Material
  float3 emission;
  float3 color;
};

struct Scene {
  int numObjects;
  Sphere* objects;
};

struct HitData {
  float t;
  // index of object hit
  int index;
};

struct Ray {
  float3 origin;
  float3 direction;
};

__device__ bool intersectSphere(const Ray& ray, const Sphere& sphere, float* t) {
	float3 offset = ray.origin - sphere.pos;
	float a = dot(ray.direction, ray.direction);
	float b = 2.0 * dot(ray.direction, offset);
	float c = dot(offset, offset) - sphere.radius * sphere.radius;
	float determinant = b*b - 4 * a*c;
	// hit one or both sides
	if (determinant >= 0) {
		float tNear = (-b - sqrt((b*b) - 4.0*a*c))/(2.0*a);
		float tFar = (-b + sqrt((b*b) - 4.0*a*c))/(2.0*a);
		if(tNear > 0 && tFar > 0)
			*t = min(tNear, tFar);
		else if(tNear > 0)
			*t = tNear;
		else
			*t = tFar;
		return true;
	}
	return false;
}

__device__ bool intersectScene(const Scene& scene, const Ray& ray, HitData* hitData) {
  float tNearest = 1000000.0f;
	float t = 0;
	bool hit = false;
	for (int i = 0; i < scene.numObjects; i++) {
		//if there was an intersection and it is the closest
		if (intersectSphere(ray, scene.objects[i], &t) && t > 0 && t < tNearest) {
			tNearest = t;
			hit = true;
			hitData->t = t;
			hitData->index = i;
		}
	}
	return hit;
}

__device__ float3 orthoVector(float3 v) {
    //  See : http://lolengine.net/blog/2013/09/21/picking-orthogonal-vector-combing-coconuts
    return (abs(v.x) > abs(v.z)) ? make_float3(-v.y, v.x, 0.0f)  : make_float3(0.0f, -v.z, v.y);
}

__device__ float3 getCosineWeightedNormal(float3 dir, hiprandState* randState) {
  float power = 1.0f; //0 for unbiased
  dir = normalize(dir);
	float3 o1 = normalize(orthoVector(dir));
	float3 o2 = normalize(cross(dir, o1));
	float2 r = make_float2(hiprand_uniform(randState), hiprand_uniform(randState));
	r.x = r.x * 2.0f * HIP_PI_F;
	r.y = pow(r.y, 1.0f / (power + 1.0f));
	float oneminus = sqrt(1.0 - r.y * r.y);
	return cos(r.x) * oneminus * o1 + sin(r.x) * oneminus * o2 + r.y * dir;
}

__device__ float3 trace_ray(const Scene& scene, Ray ray, hiprandState* randState) {
  HitData hitData;
  float3 color = make_float3(0,0,0);
  float3 mask = make_float3(1,1,1);
  
  for (int n = 0; n < MAX_BOUNCES; n++) {
    // ray leaves the scene
    if (!intersectScene(scene, ray, &hitData))
      return color;
    
    // accumulate emmission
    color += mask * scene.objects[hitData.index].emission;
    // attenuate color for next bounce
    mask *= scene.objects[hitData.index].color; //account for incoming direction??

    // bounce off surface
    float3 pos = ray.origin + ray.direction * hitData.t;
    float3 normal = normalize(pos - scene.objects[hitData.index].pos);
    // flip normal if necessary
    normal = dot(normal, ray.direction) < 0 ? normal : -1 * normal;
    // create next ray
    ray.origin = pos + normal * PUSH_RAY_ORIGIN;
    ray.direction = normalize(getCosineWeightedNormal(normal, randState));
  }
  return color;
}

__global__ void pixel_kernel(float* output, hiprandState* randStates, Scene scene, float3* rayBasis, float3* eyePos, int spp) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int id = x * SCREEN_W + y;

  if (x >= SCREEN_W || y >= SCREEN_H)
    return;
  
  //copy random state to local memory
  hiprandState localRandState = randStates[id];
  
  // take samples
  float3 color = make_float3(0.0f, 0.0f, 0.0f);
  for (int i = 0; i < spp; i++) {
    // determine ray direction by interpolating from basis
    float2 screenPos = make_float2(x + hiprand_uniform(&localRandState)*2.0f - 1.0f, y + hiprand_uniform(&localRandState)*2.0f - 1.0f);
    screenPos /= make_float2(SCREEN_W, SCREEN_H);
    Ray ray;
    ray.origin = *eyePos;
    ray.direction = lerp(lerp(rayBasis[0], rayBasis[1], screenPos.y), lerp(rayBasis[2], rayBasis[3], screenPos.y), 1.0f-screenPos.x);
    // trace ray and accumulate color
    color += trace_ray(scene, ray, &localRandState);
  }
  color /= (float)spp;
  
  // write to output buffer
  output[x*SCREEN_W*NUM_CHANNELS + y*NUM_CHANNELS + 0] = color.x;
  output[x*SCREEN_W*NUM_CHANNELS + y*NUM_CHANNELS + 1] = color.y;
  output[x*SCREEN_W*NUM_CHANNELS + y*NUM_CHANNELS + 2] = color.z;
  // copy rand state back to global memory
  randStates[id] = localRandState;
}

__global__ void setup_random(hiprandState* states) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int id = x * SCREEN_W + y;
  if (x >= SCREEN_W || y >= SCREEN_H)
    return;
  hiprand_init(id, 0, 0, &states[id]);
}


int main(int argc, char** argv) {
  // determine how to distribute work to GPU
  int blockSize = 32;
  int bx = (SCREEN_W + blockSize - 1)/blockSize;
  int by = (SCREEN_H + blockSize - 1)/blockSize;
  dim3 gridSize = dim3(bx, by);
  dim3 dimBlock = dim3(blockSize, blockSize);

  std::cout << gridSize.x << ", " << gridSize.y << std::endl;

  gpuErrchk(hipSetDevice(1));
  
  // random number generator states: 1 for each pixel/thread
  hiprandState* d_states;
  int numCurandStates = SCREEN_W*SCREEN_H;
  gpuErrchk(hipMalloc(&d_states, numCurandStates * sizeof(hiprandState)));
  setup_random<<<gridSize, dimBlock>>>(d_states);
  
  // create scene
  Scene d_scene;
  d_scene.numObjects = 9;
  Sphere spheres[] = {
   { 1e5f, { 1e5f + 1.0f, 40.8f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { 0.75f, 0.25f, 0.25f } }, //Left 
   { 1e5f, { -1e5f + 99.0f, 40.8f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { .25f, .25f, .75f } }, //Right 
   { 1e5f, { 50.0f, 40.8f, 1e5f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f } }, //Back 
   { 1e5f, { 50.0f, 40.8f, -1e5f + 600.0f }, { 0.0f, 0.0f, 0.0f }, { 1.00f, 1.00f, 1.00f } }, //Frnt 
   { 1e5f, { 50.0f, 1e5f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f } }, //Botm 
   { 1e5f, { 50.0f, -1e5f + 81.6f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f } }, //Top 
   { 16.5f, { 27.0f, 16.5f, 47.0f }, { 0.0f, 0.0f, 0.0f }, { 1.0f, 1.0f, 1.0f } }, // small sphere 1
   { 16.5f, { 73.0f, 16.5f, 78.0f }, { 0.0f, 0.0f, 0.0f }, { 1.0f, 1.0f, 1.0f } }, // small sphere 2
   { 600.0f, { 50.0f, 681.6f - .77f, 81.6f }, { 2.0f, 1.8f, 1.6f }, { 0.0f, 0.0f, 0.0f } }  // Light
  };
  gpuErrchk(hipMalloc(&d_scene.objects, d_scene.numObjects*sizeof(Sphere)));
  gpuErrchk(hipMemcpy(d_scene.objects, spheres, d_scene.numObjects*sizeof(Sphere), hipMemcpyHostToDevice));

  // create camera and compute eye ray basis
  Camera camera(glm::vec3(50, 52, 295.6));
  float3 eyeRayBasis[4];
  camera.getEyeRayBasis(eyeRayBasis, SCREEN_W, SCREEN_H);
  float3* d_eyeRayBasis;
  gpuErrchk(hipMalloc(&d_eyeRayBasis, 4*sizeof(float3)));
  gpuErrchk(hipMemcpy(d_eyeRayBasis, eyeRayBasis, 4*sizeof(float3), hipMemcpyHostToDevice));
  float3* d_eyePos;
  gpuErrchk(hipMalloc(&d_eyePos, sizeof(float3)));
  gpuErrchk(hipMemcpy(d_eyePos, &camera.Position, sizeof(float3), hipMemcpyHostToDevice));
  
  // allocate output buffer on host and device
  float* screenBuffer = new float[SCREEN_W*SCREEN_H*NUM_CHANNELS];
  float* d_screenBuffer;
  gpuErrchk(hipMalloc(&d_screenBuffer, SCREEN_W*SCREEN_H*NUM_CHANNELS*sizeof(float)));

  //measure how long kernel takes
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // run kernel
  hipEventRecord(start);
  pixel_kernel<<<gridSize, dimBlock>>>(d_screenBuffer, d_states, d_scene, d_eyeRayBasis, d_eyePos, SAMPLES);
  hipEventRecord(stop);

  // copy output buffer back to host
  gpuErrchk(hipMemcpy(screenBuffer, d_screenBuffer, SCREEN_W*SCREEN_H*NUM_CHANNELS*sizeof(float), hipMemcpyDeviceToHost));

  // save bitmap
  unsigned char* outBuffer = new unsigned char[SCREEN_W*SCREEN_H*3];
  for (int i = 0; i < SCREEN_W*SCREEN_H*3; i++)
    outBuffer[i] = (unsigned char)min(255, max(0, (int)(255.0f * screenBuffer[i])));

  stbi_write_bmp("output.bmp", SCREEN_W, SCREEN_H, 3, outBuffer);
  delete[] outBuffer;
  
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Kernel took %fms (%f fps)\n", milliseconds, 1000.0f/milliseconds);

  // clean up
  hipFree(d_screenBuffer);
  delete[] screenBuffer;
  return 0;
}
