#include "hip/hip_runtime.h"
#include "pathtrace.h"
#include <hip/hip_vector_types.h>
#include <hip/hip_math_constants.h>


// Renderer constants
#define MAX_BOUNCES 5
#define PUSH_RAY_ORIGIN 0.05f

struct HitData {
  float t;
  // index of object hit
  int index;
};

struct Ray {
  float3 origin;
  float3 direction;
};

enum Features {COLOR=0, NORMAL=1, ALBEDO=2, DEPTH=3, NUM_FEATURES=4};

// Output of a single ray trace
struct TraceOutput {
  float3 color;
  float3 normal;
  float3 albedo;
  float depth;

  __device__ TraceOutput() {
    color = make_float3(0.0f, 0.0f, 0.0f);
    normal = make_float3(0.0f, 0.0f, 0.0f);
    albedo = make_float3(0.0f, 0.0f, 0.0f);
    depth = 0.0f;
  }
};

// Compute variance for features
struct OnlineVarianceBuffer {
  int n[NUM_FEATURES];
  float mean[NUM_FEATURES];
  float M2[NUM_FEATURES];

  __device__ OnlineVarianceBuffer() {
    for(int i = 0; i < NUM_FEATURES; i++) {
      n[i] = 0;
      mean[i] = 0.0f;
      M2[i] = 0.0f;
    }
  }

  __device__ void updateVariance(float x, Features feature) {
    n[feature] += 1;
    float delta = x - mean[feature];
    mean[feature] += delta/n[feature];
    float delta2 = x - mean[feature];
    M2[feature] += delta*delta2;
  }

  __device__ float getVariance(Features feature) {
    if(n[feature] < 2)
      return 0.0f;
    return (M2[feature] / (n[feature]-1));
  }
};

__device__ float luminance(float3 color) {
  return 0.2126*color.x + 0.7152*color.y + 0.0722*color.z;
}


__device__ bool intersectSphere(const Ray& ray, const Sphere& sphere, float* t) {
	float3 offset = ray.origin - sphere.pos;
	float a = dot(ray.direction, ray.direction);
	float b = 2.0 * dot(ray.direction, offset);
	float c = dot(offset, offset) - sphere.radius * sphere.radius;
	float determinant = b*b - 4 * a*c;
	// hit one or both sides
	if (determinant >= 0) {
		float tNear = (-b - sqrt((b*b) - 4.0*a*c))/(2.0*a);
		float tFar = (-b + sqrt((b*b) - 4.0*a*c))/(2.0*a);
		if(tNear > 0 && tFar > 0)
			*t = min(tNear, tFar);
		else if(tNear > 0)
			*t = tNear;
		else
			*t = tFar;
		return true;
	}
	return false;
}

__device__ bool intersectScene(const Scene& scene, const Ray& ray, HitData* hitData) {
  float tNearest = 1000000.0f;
	float t = 0;
	bool hit = false;
	for (int i = 0; i < scene.numObjects; i++) {
		//if there was an intersection and it is the closest
		if (intersectSphere(ray, scene.objects[i], &t) && t > 0 && t < tNearest) {
			tNearest = t;
			hit = true;
			hitData->t = t;
			hitData->index = i;
		}
	}
	return hit;
}

__device__ float testShadowRay(const Scene& scene, const Ray& ray, int lightIndex) {
  float tNearest = length(scene.objects[lightIndex].pos- make_float3(0, scene.objects[lightIndex].radius, 0) - ray.origin);
	float t = 0;
	for (int i = 0; i < scene.numObjects; i++) {
		//if there was an intersection and it is the closest
		if (i != lightIndex && intersectSphere(ray, scene.objects[i], &t) && t > 0 && t < tNearest) {
			return 0.0f;
		}
	}
	return 1.0f;
}

__device__ float3 orthoVector(float3 v) {
    //  See : http://lolengine.net/blog/2013/09/21/picking-orthogonal-vector-combing-coconuts
    return (abs(v.x) > abs(v.z)) ? make_float3(-v.y, v.x, 0.0f)  : make_float3(0.0f, -v.z, v.y);
}

__device__ float3 getCosineWeightedNormal(float3 dir, hiprandState* randState) {
  float power = 1.0f; //0 for unbiased
  dir = normalize(dir);
	float3 o1 = normalize(orthoVector(dir));
	float3 o2 = normalize(cross(dir, o1));
	float2 r = make_float2(hiprand_uniform(randState), hiprand_uniform(randState));
	r.x = r.x * 2.0f * HIP_PI_F;
	r.y = pow(r.y, 1.0f / (power + 1.0f));
	float oneminus = sqrt(1.0 - r.y * r.y);
	return cos(r.x) * oneminus * o1 + sin(r.x) * oneminus * o2 + r.y * dir;
}

__device__ float3 getDirectLighting(const Scene& scene, const float3& normal, const float3& position) {
  // TODO: sample random point on light
  // TODO: make lights array
  int lightIndex = 8;
  float3 lightDir = normalize(scene.objects[lightIndex].pos - make_float3(0, scene.objects[lightIndex].radius, 0) - position);
  float diffuse = clamp(dot(lightDir, normal), 0.0, 1.0);
  Ray shadowRay;
  shadowRay.origin = position + normal * PUSH_RAY_ORIGIN;
  shadowRay.direction = lightDir;
  return diffuse * scene.objects[lightIndex].emission * testShadowRay(scene, shadowRay, lightIndex);
}

__device__ void trace_ray(TraceOutput& L, const Scene& scene, Ray ray, hiprandState* randState, int x, int y, OnlineVarianceBuffer& var) {
  HitData hitData;
  float3 color = make_float3(0,0,0);
  float3 mask = make_float3(1,1,1);
  
  for (int n = 0; n < MAX_BOUNCES; n++) {
    // intersect with surface
    if (!intersectScene(scene, ray, &hitData)) {
      // ray leaves the scene
      L.color += color;
      return;
    }
    // get intersection position and normal
    float3 pos = ray.origin + ray.direction * hitData.t;
    float3 normal = normalize(pos - scene.objects[hitData.index].pos);
    // flip normal if necessary
    normal = dot(normal, ray.direction) < 0 ? normal : -1 * normal;
    
    // direct lighting
    //color += mask * getDirectLighting(scene, normal, pos) * scene.objects[hitData.index].color * 0.5f;
    // hack for aliasing issue with bright light source
    if (n == 0)
      color += clamp(mask * scene.objects[hitData.index].emission, 0.0, 1.0);
    else
      color += mask * scene.objects[hitData.index].emission;
    mask *= scene.objects[hitData.index].color;
    
    // create next ray
    ray.origin = pos + normal * PUSH_RAY_ORIGIN;
    // diffuse bounce
    ray.direction = normalize(getCosineWeightedNormal(normal, randState));
    // makeshift glossy BRDF
    //ray.direction = reflect(ray.direction, normal);
    //ray.direction += 0.01* make_float3(hiprand_uniform(randState), hiprand_uniform(randState), hiprand_uniform(randState)) - 0.005;
    //ay.direction = normalize(ray.direction);

    // record first bounce information
    if(n == 0) {
      L.normal += normal;
      L.albedo += scene.objects[hitData.index].color;
      L.depth += hitData.t;
      // update variances
      var.updateVariance(luminance(normal), NORMAL);
      var.updateVariance(luminance(scene.objects[hitData.index].color), ALBEDO);
      var.updateVariance(hitData.t, DEPTH);
    }
  }

  L.color += color;
  // update color variance with final sample color
  var.updateVariance(luminance(color), COLOR);
}

__global__ void pixel_kernel(OutputBuffer output, hiprandState* randStates, Scene scene, float3* rayBasis, float3* eyePos, int spp) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int id = x * output.width + y;

  if (x >= output.width || y >= output.height)
    return;
  
  //copy random state to local memory
  hiprandState localRandState = randStates[id];
  
  // take samples
  float3 color = make_float3(0.0f, 0.0f, 0.0f);
  OnlineVarianceBuffer var;
  TraceOutput L;

  for (int i = 0; i < spp; i++) {
    // determine ray direction by interpolating from basis
    float2 screenPos = make_float2(x, y);
    if (spp != 1) {
      screenPos.x += hiprand_uniform(&localRandState)*1.0f - 0.5f;
      screenPos.y += hiprand_uniform(&localRandState)*1.0f - 0.5f;
    }
    screenPos /= make_float2(output.width, output.height);
    Ray ray;
    ray.origin = *eyePos;
    ray.direction = lerp(lerp(rayBasis[0], rayBasis[1], screenPos.y), lerp(rayBasis[2], rayBasis[3], screenPos.y), 1.0f-screenPos.x);
    // trace ray and accumulate color
    trace_ray(L, scene, ray, &localRandState, x, y, var);
  }
  // average over all samples
  L.color /= (float)spp;
  L.normal /= (float)spp;
  L.albedo /= (float)spp;
  L.depth /= (float)spp;
  
  // write to output buffer
  output.buffer[x*output.width*14 + y*14 + 0] = L.color.x;
  output.buffer[x*output.width*14 + y*14 + 1] = L.color.y;
  output.buffer[x*output.width*14 + y*14 + 2] = L.color.z;
  output.buffer[x*output.width*14 + y*14 + 3] = L.normal.x;
  output.buffer[x*output.width*14 + y*14 + 4] = L.normal.y;
  output.buffer[x*output.width*14 + y*14 + 5] = L.normal.z;
  output.buffer[x*output.width*14 + y*14 + 6] = L.albedo.x;
  output.buffer[x*output.width*14 + y*14 + 7] = L.albedo.y;
  output.buffer[x*output.width*14 + y*14 + 8] = L.albedo.z;
  output.buffer[x*output.width*14 + y*14 + 9] = L.depth;
  // get final variances
  output.buffer[x*output.width*14 + y*14 + 10] = var.getVariance(COLOR);
  output.buffer[x*output.width*14 + y*14 + 11] = var.getVariance(NORMAL);
  output.buffer[x*output.width*14 + y*14 + 12] = var.getVariance(ALBEDO);
  output.buffer[x*output.width*14 + y*14 + 13] = var.getVariance(DEPTH);
  // copy rand state back to global memory
  randStates[id] = localRandState;
}

__global__ void setup_random(hiprandState* states, int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int id = x * width + y;
  if (x >= width || y >= height)
    return;
  hiprand_init(id, 0, 0, &states[id]);
}