#include "Renderer.h"
#include "Denoiser.h"
#include "Camera.h"
#include "Window.h"
#include "Scene.h"
#include <iostream>
#include <string>
#include "args.hxx"

#define TINYEXR_IMPLEMENTATION
#include "tinyexr.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"


#include <boost/filesystem.hpp>
#include <boost/python.hpp>

namespace py = boost::python;
namespace fs = boost::filesystem;

struct PythonState {
    py::object main_module;
    py::object globals;

    PythonState()
        : main_module(py::object(
              py::handle<>(py::borrowed(PyImport_AddModule("__main__")))))
    {
        globals = main_module.attr("__dict__");
    }

    py::object import(const std::string& module_path)
    {
        return _import(fs::path(module_path));
    }

    py::object _import(const fs::path& module_path)
    {
        try {
            py::dict locals;
            locals["mname"] = module_path.stem().string();
            locals["filename"] = module_path.string();
            py::exec("import importlib.util\n"
                     "spec = importlib.util.spec_from_file_location(mname, "
                     "filename)\n"
                     "imported = importlib.util.module_from_spec(spec)\n"
                     "spec.loader.exec_module(imported)",
                globals, locals);
            return locals["imported"];
        } catch (py::error_already_set& err) {
            PyErr_Print();
        }
        return py::object();
    }

    py::object exec(const char* code, py::dict& locals)
    {
        try {
            return py::exec(code, globals, locals);
        } catch (py::error_already_set& err) {
            PyErr_Print();
        }
        return py::object();
    }

    py::object exec(const char* code)
    {
        try {
            return py::exec(code, globals, globals);
        } catch (py::error_already_set& err) {
            PyErr_Print();
        }
        return py::object();
    }
};


int main(int argc, const char** argv) {
  // set up argument parser
  args::ArgumentParser parser("cuda-pathtrace");
  parser.LongSeparator(" ");
  args::HelpFlag help(parser, "help", "Display this help menu", {'h', "help"});
  args::ValueFlag<int> argWidth(parser, "w", "Image/window width (default 512)", {'w', "width"});
  args::ValueFlag<int> argSamples(parser, "samples", "Number of samples per pixel (default 4)", {'s', "samples"});
  args::ValueFlag<int> argDevice(parser, "device", "Which CUDA device to use (default 0)", {'d', "device"});
  args::ValueFlag<int> argThreads(parser, "threads", "Number of threads per block (default 8)", {'t', "threads-per-block"});
  args::ValueFlag<float> argCameraX(parser, "x", "Starting camera position x", {'x', "camera-x"});
  args::ValueFlag<float> argCameraY(parser, "y", "Starting camera position y", {'y', "camera-y"});
  args::ValueFlag<float> argCameraZ(parser, "z", "Starting camera position z", {'z', "camera-z"});
  args::ValueFlag<float> argViewYaw(parser, "yaw", "Starting camera view yaw", {'c', "camera-yaw"});
  args::ValueFlag<float> argViewPitch(parser, "pitch", "Starting camera view pitch", {'p', "camera-pitch"});
  args::ValueFlag<std::string> argOutput(parser, "path", "Prefix of output file name(s) (default output/output)", {'o', "output"});
  args::Flag argNoBitmaps(parser, "nobitmap", "Do not output bitmap features - only the exr", {'n', "nobitmap"});
  args::Flag argInteractive(parser, "interactive", "Open in interactive mode  - will only render a single frame if not set", {'i', "interactive"});
  try {
    parser.ParseCLI(argc, argv);
  }
  catch (args::Help) {
    std::cout << parser;
    return 0;
  }
  catch (args::ParseError e) {
    std::cerr << e.what() << std::endl;
    std::cerr << parser;
    return 1;
  }
  catch (args::ValidationError e) {
    std::cerr << e.what() << std::endl;
    std::cerr << parser;
    return 1;
  }
  //get arguments
  int width = (argWidth) ? args::get(argWidth) : 512;
  int height = width;
  int threadsPerBlock = (argThreads) ? args::get(argThreads) : 8;
  int samplesPerPixel = (argSamples) ? args::get(argSamples) : 4;
  int cudaDevice = (argDevice) ? args::get(argDevice) : 0;
  // camera arguments
  float cameraPos[3] = { 50.0f, 52.0f, 295.6f };
  if (argCameraX) 
    cameraPos[0] = args::get(argCameraX);
  if (argCameraY) 
    cameraPos[1] = args::get(argCameraY);
  if (argCameraZ) 
    cameraPos[2] = args::get(argCameraZ);
  float cameraView[2] = {-90.0f, 0.0f};
  if (argViewYaw)
    cameraView[0] = args::get(argViewYaw);
  if (argViewPitch)
    cameraView[1] = args::get(argViewPitch);
  bool denoising = true;
  std::string outputName = (argOutput) ? args::get(argOutput) : "output/out";
  std::cout << "cuda-pathtrace 0.2" << std::endl;
  std::cout << "------------------" << std::endl;
  std::cout << "Dimensions: " << width << " x " << height << std::endl;
  std::cout << "Threads per block: " << threadsPerBlock << std::endl;
  std::cout << "Samples per pixel: " << samplesPerPixel << std::endl;
  std::cout << "Using CUDA device: " << cudaDevice << std::endl;
  if (!argInteractive)
    std::cout << "Output file prefix: " << outputName << std::endl;
  std::cout << "Camera: " << cameraPos[0] << " " << cameraPos[1] << " " << cameraPos[2] << " " << cameraView[0] << " " << cameraView[1] << std::endl;

  // set cuda device
  gpuErrchk(hipSetDevice(cudaDevice));
  //if (argIteractive)
  //   gpuErrchk(cudaGLSetGLDevice(cudaDevice));

  Py_Initialize();
  PyEval_InitThreads();
  long _tensor_ptr = -1;
  py::dict locals;
  PythonState state;

  state.exec("import torch\n"
              "def make_tensor():\n"
              "    return torch.cuda.FloatTensor(512, 512, 14)\n"
              "def modify_tensor(tensor):\n"
              "    tensor[:, :, 0] = 1.0");

  py::object torch = state.globals["torch"];

  py::object make_tensor = state.globals["make_tensor"];
  py::object modify_tensor = state.globals["modify_tensor"];

  py::object tensor = make_tensor();

  locals["t"] = tensor;

  state.exec("data_ptr = t.data_ptr()", locals);

  _tensor_ptr = py::extract<long>(locals["data_ptr"]);

  std::cout << "init torch.cuda.FloatTensor=" << _tensor_ptr
            << std::endl;

  if (_tensor_ptr < 0) {
      return 0;
  }

  void* tensor_ptr = reinterpret_cast<void*>(_tensor_ptr);

  // load scene and create renderer
  Scene scene;
  Renderer renderer(width, height, samplesPerPixel, threadsPerBlock);
  Denoiser denoiser(width, height, threadsPerBlock);
  Camera camera(glm::make_vec3(cameraPos), cameraView[0], cameraView[1]);
  
  // allocate output buffer
  OutputBuffer d_buffer(width, height);
  if (true) {
    // torch tensor
    d_buffer.buffer = (float*)tensor_ptr;
  }
  else {
    // regular cuda memory
    d_buffer.AllocateGPU();
  }

  if (argInteractive) {
    // interactive (realtime) mode
    Window window(width, height, &camera);
    GLPixelBuffer denoisedBuffer(width, height);

    while(!window.ShouldClose()) {
      window.DoMovement();
      renderer.Render(d_buffer, scene, camera);
      modify_tensor(tensor);
      denoiser.Denoise(d_buffer, denoisedBuffer);
      window.DrawToScreen(denoisedBuffer);
    }
  }
  else {
    // data collection (single frame render) mode
    // render frame
    float renderTime = renderer.Render(d_buffer, scene, camera);
    std::cout << "Render completed in " << renderTime << "ms (" << 1000.0f/renderTime << " fps)" << std::endl;
    std::cout << std::endl;
    // save results
    OutputBuffer buffer(width, height);
    buffer.AllocateCPU();
    buffer.CopyFromGPU(d_buffer);
    buffer.SaveEXR(outputName+".exr");
    if(!argNoBitmaps)
      buffer.SaveBitmaps(outputName);
    buffer.FreeCPU();
  }
  
  return 0;
}